/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "weightOnlyBatchedGemv/common.h"
#include "weightOnlyBatchedGemv/utility.h"

namespace tensorrt_llm
{
namespace kernels
{
template <WeightOnlyQuantType QType, typename WeightOnlyFlag, template <typename T> class ActOp, bool Zero, bool Bias,
    int N_PER_BLOCK, int BATCH, int BLOCK_SIZE>
struct WeightOnlyBatchedGemvKernelLauncher
{
    static void run(const WeightOnlyParams& params, hipStream_t stream);
};

template <WeightOnlyQuantType QType, typename WeightOnlyFlag, template <typename T> class ActOp, int N_PER_BLOCK,
    int BATCH, int BLOCK_SIZE>
void select_zero_bias(const WeightOnlyParams& params, hipStream_t stream)
{
    if (params.zeros && params.bias)
    {
        WeightOnlyBatchedGemvKernelLauncher<QType, WeightOnlyFlag, ActOp, true, true, N_PER_BLOCK, BATCH,
            BLOCK_SIZE>::run(params, stream);
    }
    else if (params.zeros && !params.bias)
    {
        WeightOnlyBatchedGemvKernelLauncher<QType, WeightOnlyFlag, ActOp, true, false, N_PER_BLOCK, BATCH,
            BLOCK_SIZE>::run(params, stream);
    }
    else if (!params.zeros && params.bias)
    {
        WeightOnlyBatchedGemvKernelLauncher<QType, WeightOnlyFlag, ActOp, false, true, N_PER_BLOCK, BATCH,
            BLOCK_SIZE>::run(params, stream);
    }
    else
    {
        WeightOnlyBatchedGemvKernelLauncher<QType, WeightOnlyFlag, ActOp, false, false, N_PER_BLOCK, BATCH,
            BLOCK_SIZE>::run(params, stream);
    }
}

template <WeightOnlyQuantType QType, typename WeightOnlyFlag, int N_PER_BLOCK, int BATCH, int BLOCK_SIZE>
void select_activation(WeightOnlyActivationType atype, const WeightOnlyParams& params, hipStream_t stream)
{
    switch (atype)
    {
    case WeightOnlyActivationType::Gelu:
    {
        select_zero_bias<QType, WeightOnlyFlag, GeluActivation, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
        break;
    }
    case WeightOnlyActivationType::Relu:
    {
        select_zero_bias<QType, WeightOnlyFlag, ReluActivation, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
        break;
    }
    case WeightOnlyActivationType::Identity:
    {
        select_zero_bias<QType, WeightOnlyFlag, IdentityActivation, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
        break;
    }
    default:
    {
        throw std::runtime_error("Use unsupported activation");
        break;
    }
    }
}

template <typename WeightOnlyFlag, int N_PER_BLOCK, int BATCH, int BLOCK_SIZE>
void select_quant_type(
    WeightOnlyQuantType qtype, WeightOnlyActivationType atype, const WeightOnlyParams& params, hipStream_t stream)
{
    if (qtype == WeightOnlyQuantType::Int4b)
    {
        select_activation<WeightOnlyQuantType::Int4b, WeightOnlyFlag, N_PER_BLOCK, BATCH, BLOCK_SIZE>(
            atype, params, stream);
    }
    else if (qtype == WeightOnlyQuantType::Int8b)
    {
        select_activation<WeightOnlyQuantType::Int8b, WeightOnlyFlag, N_PER_BLOCK, BATCH, BLOCK_SIZE>(
            atype, params, stream);
    }
    else
    {
        throw std::runtime_error("Unknown QuantType");
    }
}

template <int N_PER_BLOCK, int BATCH, int BLOCK_SIZE>
void select_groupwise_weight_only(WeightOnlyQuantType qtype, WeightOnlyType wtype, WeightOnlyActivationType atype,
    const WeightOnlyParams& params, hipStream_t stream)
{
    if (wtype == WeightOnlyType::GroupWise && params.group_size == 64)
    {
        select_quant_type<WeightOnlyGroupWise<64>, N_PER_BLOCK, BATCH, BLOCK_SIZE>(qtype, atype, params, stream);
    }
    else if (wtype == WeightOnlyType::GroupWise && params.group_size == 128)
    {
        select_quant_type<WeightOnlyGroupWise<128>, N_PER_BLOCK, BATCH, BLOCK_SIZE>(qtype, atype, params, stream);
    }
    else
    {
        throw std::runtime_error("Only support groupwise weight only for gs=64/128");
    }
}

void weight_only_batched_gemv_launcher(WeightOnlyQuantType qtype, WeightOnlyType wtype, WeightOnlyActivationType atype,
    const WeightOnlyParams& params, hipStream_t stream)
{
    if (wtype == WeightOnlyType::PerChannel)
    {
        if (qtype == WeightOnlyQuantType::Int4b)
        {
            switch (params.m)
            {
            case 1:
            {
                select_activation<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel, 1, 1, 192>(atype, params, stream);
                break;
            }
            case 2:
            {
                select_activation<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel, 2, 2, 128>(atype, params, stream);
                break;
            }
            case 3:
            {
                select_activation<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel, 2, 3, 256>(atype, params, stream);
                break;
            }
            case 4:
            {
                select_activation<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel, 4, 4, 256>(atype, params, stream);
                break;
            }
            default:
            {
                throw std::runtime_error("Weight only cuda kernel only supported bs <= 4");
                break;
            }
            }
        }
        else if (qtype == WeightOnlyQuantType::Int8b)
        {
            switch (params.m)
            {
            case 1:
            {
                select_activation<WeightOnlyQuantType::Int8b, WeightOnlyPerChannel, 2, 1, 256>(atype, params, stream);
                break;
            }
            case 2:
            {
                select_activation<WeightOnlyQuantType::Int8b, WeightOnlyPerChannel, 2, 2, 256>(atype, params, stream);
                break;
            }
            case 3:
            {
                select_activation<WeightOnlyQuantType::Int8b, WeightOnlyPerChannel, 2, 3, 256>(atype, params, stream);
                break;
            }
            case 4:
            {
                select_activation<WeightOnlyQuantType::Int8b, WeightOnlyPerChannel, 2, 4, 256>(atype, params, stream);
                break;
            }
            default:
            {
                throw std::runtime_error("Weight only cuda kernel only supported bs <= 4");
                break;
            }
            }
        }
    }
    else if (wtype == WeightOnlyType::GroupWise)
    {
        switch (params.m)
        {
        case 1:
        {
            select_groupwise_weight_only<2, 1, 256>(qtype, wtype, atype, params, stream);
            break;
        }
        case 2:
        {
            select_groupwise_weight_only<2, 2, 256>(qtype, wtype, atype, params, stream);
            break;
        }
        case 3:
        {
            select_groupwise_weight_only<2, 3, 128>(qtype, wtype, atype, params, stream);
            break;
        }
        case 4:
        {
            select_groupwise_weight_only<2, 4, 128>(qtype, wtype, atype, params, stream);
            break;
        }
        default:
        {
            throw std::runtime_error("Weight only cuda kernel only supported bs <= 4");
            break;
        }
        }
    }
}
} // namespace kernels
} // namespace tensorrt_llm
